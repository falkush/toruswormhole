#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>


static uint8_t* buffer=0;
static double* vecl=0;
static uint8_t* stars = 0;

const static int starsize = 100;


__device__ void tormat(double phi, double theta, double* mat)
{
	mat[0] = cos(theta) * sin(phi);
	mat[3] = cos(theta) * cos(phi);
	mat[6] = sin(theta);

	mat[2] = cos(phi);
	mat[5] = -sin(phi);
	mat[8] = 0;

	mat[1] = -sin(theta) * sin(phi);
	mat[4] = -sin(theta) * cos(phi);
	mat[7] = cos(theta);
}

__device__ double matdet(double* m)
{
	return m[0] * (m[4]*m[8]-m[5]*m[7]) - m[1] * (m[3]*m[8]-m[5]*m[6]) + m[2] * (m[3]*m[7]-m[4]*m[6]);
}

__device__ void matinv(double* m, double* res)
{
	res[0] = m[4] * m[8] - m[5] * m[7];
	res[1] = m[2] * m[7] - m[1] * m[8];
	res[2] = m[1] * m[5] - m[2] * m[4];
	res[3] = m[5] * m[6] - m[3] * m[8];
	res[4] = m[0] * m[8] - m[2] * m[6];
	res[6] = m[3] * m[7] - m[4] * m[6];
	res[5] = m[2] * m[3] - m[0] * m[5];
	res[7] = m[1] * m[6] - m[0] * m[7];
	res[8] = m[0] * m[4] - m[1] * m[3];
}

__device__ void matmult(double* m1, double* m2, double* res)
{
	res[0] = m1[0] * m2[0] + m1[1] * m2[3] + m1[2] * m2[6];
	res[1] = m1[0] * m2[1] + m1[1] * m2[4] + m1[2] * m2[7];
	res[2] = m1[0] * m2[2] + m1[1] * m2[5] + m1[2] * m2[8];
	res[3] = m1[3] * m2[0] + m1[4] * m2[3] + m1[5] * m2[6];
	res[4] = m1[3] * m2[1] + m1[4] * m2[4] + m1[5] * m2[7];
	res[5] = m1[3] * m2[2] + m1[4] * m2[5] + m1[5] * m2[8];
	res[6] = m1[6] * m2[0] + m1[7] * m2[3] + m1[8] * m2[6];
	res[7] = m1[6] * m2[1] + m1[7] * m2[4] + m1[8] * m2[7];
	res[8] = m1[6] * m2[2] + m1[7] * m2[5] + m1[8] * m2[8];
}

__device__ void matact(double* m, double vecn0, double vecn1, double vecn2, double* nvecn)
{
	nvecn[0] = m[0] * vecn0 + m[1] * vecn1 + m[2] * vecn2;
	nvecn[1] = m[3] * vecn0 + m[4] * vecn1 + m[5] * vecn2;
	nvecn[2] = m[6] * vecn0 + m[7] * vecn1 + m[8] * vecn2;
}

__device__ void matflip(double* m, double* res)
{
	res[0] = m[6];
	res[1] = m[7];
	res[2] = m[8];
	res[3] = m[3];
	res[4] = m[4];
	res[5] = m[5];
	res[6] = -m[0];
	res[7] = -m[1];
	res[8] = -m[2];
}

__device__ void matflip2(double* m, double* res)
{
	res[0] = m[2];
	res[1] = m[1];
	res[2] = -m[0];
	res[3] = m[5];
	res[4] = m[4];
	res[5] = -m[3];
	res[6] = m[8];
	res[7] = m[7];
	res[8] = -m[6];
}

__device__ double solvequartic(double a0, double b0, double c0, double d0, double e0)
{
	double tmp;
	double tmin = 65536.0;
	double sint,s;
	double r1, qds, rootint;

	double b = b0 / a0;
	double c = c0 / a0;
	double d = d0 / a0;
	double e = e0 / a0;

	double c2 = c * c;
	double bd = b * d;
	double c3 = c2 * c;
	double bcd = bd * c;
	double b2 = b * b;
	double b2e = b2 * e;
	double d2 = d * d;
	double ce = c * e;
	double bc = b * c;
	double b3 = b2 * b;
	double mbd4 = (-0.25) * b;

	double t0 = c2 - 3.0 * bd + 12.0 * e;
	double t1 = 2.0 * c3 - 9.0 * bcd + 27.0 * b2e + 27.0 * d2 - 72.0 * ce;
	double p = (8.0 * c - 3.0 * b2) / 8.0;
	double q = (b3 - 4.0 * bc + 8.0 * d) / 8.0;

	double disc = t1 * t1 - 4.0 * t0 * t0 * t0;
	
	if (disc < 0)
	{
		double st0 = sqrt(t0);
		double phi = (acos(t1 / (2.0 * t0 * st0))) / 3.0;
		sint = (-2.0 / 3.0) * p + (2.0 / 3.0) * st0 * cos(phi);
	}
	else
	{
		double bigq = cbrt((t1 + sqrt(disc)) * 0.5);
		sint = (-2.0 / 3.0) * p + (1.0 / 3.0) * (bigq + t0 / bigq);
	}	

	s = sqrt(sint) * 0.5;
	
	rootint = (sint + 2.0 * p) * (-1.0);
	qds = q / s;

	r1 = rootint + qds;

	if (r1 > 0)
	{
		r1 = 0.5 * sqrt(r1);
		tmp = mbd4 - s;

		if (tmp + r1 > 0.0000001 && tmp + r1 < tmin) tmin = tmp + r1;
		if (tmp - r1 > 0.0000001 && tmp - r1 < tmin) tmin = tmp - r1;
	}

	r1 = rootint - qds;

	if (r1 > 0)
	{
		r1 = 0.5 * sqrt(r1);
		tmp = mbd4 + s;

		if (tmp + r1 > 0.0000001 && tmp + r1 < tmin) tmin = tmp + r1;
		if (tmp - r1 > 0.0000001 && tmp - r1 < tmin) tmin = tmp - r1;
	}

	return tmin;
}

__device__ double toruscoll(double a, double b, double c, double d, double e, double f, double m, double n)
{
	double t4, t3, t2, t1, t0;

	double a2 = a * a;
	double b2 = b * b;
	double c2 = c * c;
	double d2 = d * d;
	double e2 = e * e;
	double f2 = f * f;

	double ab = a * b;
	double cd = c * d;
	double ef = e * f;
	double abc = ab * c;

	double sum1 = a2 + c2 + e2;
	double sum2 = ab + cd;
	double sum3 = sum2 + ef;
	double sum4 = m + n;
	double sum5 = b2 + d2 + f2;
	double sum6 = m - n;
	double sum7 = ab + ef;

	t0 = sum5 * sum5 + sum6 * sum6;
	t0 += (-2.0) * (sum5 * sum4 - 2.0 * f2 * n);

	t1 = (b2 + d2 + f2) * sum3;
	t1 -= sum3 * sum4;
	t1 += 2.0 * ef * n;
	t1 *= 4.0;

	t2 = d * (d * (sum1 + 2.0 * c2) + 4.0 * c * sum7) + b * (b * (sum1 + 2.0 * a2) + 4.0 * a * ef) + f2 * (sum1 + 2.0 * e2);
	t2 -= sum1 * sum4;
	t2 += 2.0 * e2 * n;
	t2 *= 2.0;

	t3 = 4.0 * sum1 * sum3;

	t4 = sum1 * sum1;

	return solvequartic(t4, t3, t2, t1, t0);
}

__global__ void bufferinit(uint8_t* buffer)
{
	buffer[4 * (blockIdx.x * blockDim.x + threadIdx.x) + 3] = 255;
}

__global__ void setstars(uint8_t* stars)
{
	int i;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;

	int rand = tmp;

	for (i = 0; i < 10; i++) rand = (60493 * rand + 11) % 115249;

	if ((rand) % 5 == 0)
	{
		stars[tmp] = 255 * rand / 115249;
	}
	else
	{
		stars[tmp] = 0;
	}
}

__global__ void addKernel(uint8_t* buffer, double* vecl, double pos0,double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, bool inside, double alpha, double beta, double bigr, double r, bool other, uint8_t* stars)
{
	int i;
	double vecn0, vecn1, vecn2;
	double roomsize = 10;
	double schecker = 1;

	double geoang;
	double tcont;
	double tmpr;
	double tmpx2;
	double xyvec;

	double inv[9]{};
	double nvecn[3]{};
	double npos[3]{};
	double vl;

	double torcoll;
	double tor0, tor1, tor2;
	double theta, phi;

	double tmin,tsol;
	int tmincoord;
	double tmpsign;
	double exitalpha;

	double rayon;
	double kappa;
	double exit;
	double leangle;

	double coll0, coll1;
	int checker;
	int ctmp0, ctmp1, ctmp2;

	double u, v;
	uint8_t uv;

	int tmp2;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1920;
	int tmpy = (tmp-tmpx) /1920;

	double mat1[9]{};

	vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn0 /= vecl[tmp];
	vecn1 /= vecl[tmp];
	vecn2 /= vecl[tmp];

	if (inside)
	{
		vl = sqrt(1.0 - vecn2 * vecn2);
		geoang = atan(vecn2 / vl);

		rayon = pos2 / cos(geoang);

		kappa = sin(geoang) * rayon; //(-1)*kappa

		exitalpha = sqrt(rayon * rayon - alpha * alpha);

		if(kappa>0 && beta<rayon)
		{
			exit = sqrt(rayon * rayon - beta * beta);
			
			pos0 += (kappa-2.0*exit+exitalpha) * vecn0 / vl;
			pos1 += (kappa-2.0*exit+exitalpha) * vecn1 / vl;

			other = !other;
		}
		else
		{
			pos0 += (exitalpha + kappa) * vecn0 / vl;
			pos1 += (exitalpha + kappa) * vecn1 / vl;
		}


		vecn0 = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * vecn0 / vl;
		vecn1 = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * vecn1 / vl;
		vecn2 = -exitalpha / rayon;

		pos0 /= bigr;
		pos1 /= r;

		npos[0] = sin(pos0) * (bigr + r * cos(pos1));
		npos[1] = cos(pos0) * (bigr + r * cos(pos1));
		npos[2] = r * sin(pos1);

		tormat(pos0, pos1, mat1);
		matflip2(mat1, inv);
		matact(inv, vecn0, vecn1, vecn2, nvecn);

		vecn0 = nvecn[0];
		vecn1 = nvecn[1];
		vecn2 = nvecn[2];

		pos0 = npos[0];
		pos1 = npos[1];
		pos2 = npos[2];
	}


	torcoll = toruscoll(vecn0, pos0, vecn1, pos1, vecn2, pos2, r*r, bigr*bigr);

	for (i = 0; i < 10; i++) {
		if (torcoll != 65536)
		{
			tor0 = pos0 + torcoll * vecn0;
			tor1 = pos1 + torcoll * vecn1;
			tor2 = pos2 + torcoll * vecn2;
			xyvec = sqrt(tor0 * tor0 + tor1 * tor1);

			theta = asin(tor2 / r);
			if (xyvec < bigr) theta = M_PI - theta;
			if (theta < 0) theta += 2.0 * M_PI;

			phi = acos(tor1 / xyvec);
			if (tor0 < 0) phi *= -1.0;
			if (phi < 0) phi += 2.0 * M_PI;

			tormat(phi, theta, mat1);
			matinv(mat1, inv);
			matflip(inv, mat1);
			matact(mat1, vecn0, vecn1, vecn2, nvecn);

			npos[0] = phi * (bigr);
			npos[1] = theta * r;


			vl = sqrt(1.0 - nvecn[2] * nvecn[2]);
			geoang = atan(nvecn[2] / vl);
			rayon = alpha / cos(geoang);
			kappa = sin(geoang) * rayon; //(-1)*kappa

			if (beta < rayon)
			{
				exitalpha = sqrt(rayon * rayon - alpha * alpha);
				exit = sqrt(rayon * rayon - beta * beta);

				npos[0] += (kappa - 2.0 * exit + exitalpha) * nvecn[0] / vl;
				npos[1] += (kappa - 2.0 * exit + exitalpha) * nvecn[1] / vl;

				nvecn[0] = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * nvecn[0] / vl;
				nvecn[1] = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * nvecn[1] / vl;
				nvecn[2] = -exitalpha / rayon;

				other = !other;
			}
			else
			{
				npos[0] += 2.0 * kappa * nvecn[0] / vl;
				npos[1] += 2.0 * kappa * nvecn[1] / vl;

				nvecn[2] *= -1.0;
			}
			

			npos[0] /= bigr;
			npos[1] /= r;


			pos0 = sin(npos[0]) * (bigr + r * cos(npos[1]));
			pos1 = cos(npos[0]) * (bigr + r * cos(npos[1]));
			pos2 = r * sin(npos[1]);

			tormat(npos[0], npos[1], mat1);
			matflip2(mat1, inv);
			matact(inv, nvecn[0], nvecn[1], nvecn[2], nvecn);

			vecn0 = nvecn[0];
			vecn1 = nvecn[1];
			vecn2 = nvecn[2];

			torcoll = toruscoll(vecn0, pos0, vecn1, pos1, vecn2, pos2, r * r, bigr * bigr);
		}
	}


	if (torcoll != 65536 || isnan(vecn0))
	{
		buffer[4 * tmp] = 0;
		buffer[4 * tmp + 1] = 0;
		buffer[4 * tmp + 2] = 0;
		return;
	}

	tcont = 0;
	

	if (other)
	{
		u = starsize * ((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
		v = starsize * ((0.5 + asin(vecn2) / M_PI));

		tmp2 = (int)u + starsize * (int)v;
		//if (tmp2 < 0) uv = 0;
		//else uv = stars[tmp2];
		uv = stars[tmp2];

		if (uv % 3 == 0)
		{
			buffer[4 * tmp] = 0;
			buffer[4 * tmp + 1] = 0;
			buffer[4 * tmp + 2] = (uv * uv * uv * uv) / (255.0 * 255.0 * 255.0);
		}
		else
		{
			buffer[4 * tmp] = (uv*uv*uv) / ( 255.0*255.0);
			buffer[4 * tmp + 1] = (uv*uv*uv) / ( 255.0* 255.0);
			buffer[4 * tmp + 2] = (uv*uv*uv)/( 255.0* 255.0 );
		}
	}
	else
	{

		if (vecn0 < 0) tmpsign = -1;
		else tmpsign = 1;

		tmin = (tmpsign * roomsize - pos0) / vecn0;
		tmincoord = 0;

		if (vecn1 < 0) tmpsign = -1;
		else tmpsign = 1;
		tsol = (tmpsign * roomsize - pos1) / vecn1;
		if (tsol < tmin)
		{
			tmin = tsol;
			tmincoord = 1;
		}

		if (vecn2 < 0) tmpsign = -1;
		else tmpsign = 1;
		tsol = (tmpsign * roomsize - pos2) / vecn2;
		if (tsol < tmin)
		{
			tmin = tsol;
			tmincoord = 2;
		}

		if (tmincoord == 0)
		{
			coll0 = pos1 + tmin * vecn1;
			coll1 = pos2 + tmin * vecn2;
		}
		else if (tmincoord == 1)
		{
			coll0 = pos0 + tmin * vecn0;
			coll1 = pos2 + tmin * vecn2;
		}
		else
		{
			coll0 = pos0 + tmin * vecn0;
			coll1 = pos1 + tmin * vecn1;
		}

		checker = ((int)floor(coll0 * schecker)) % 2;
		checker += ((int)floor(coll1 * schecker)) % 2;
		if (checker < 0) checker += 2;
		checker %= 2;



		if (tmincoord == 2) {
			ctmp0 = 255;
			ctmp1 = 255;
			ctmp2 = 255;
		}
		else if (tmincoord == 0)
		{
			if (vecn0 < 0)
			{
				tmpx2 = (1.0 / 8.0) - (coll0 / (8.0 * roomsize));
				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
			else
			{
				tmpx2 = (1.0 / 2.0) + (1.0 / 8.0) + coll0 / (8.0 * roomsize);

				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
		}
		else
		{
			if (vecn1 < 0)
			{
				tmpx2 = (1.0 / 4.0) + (1.0 / 8.0) + coll0 / (8.0 * roomsize);
				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
			else
			{
				tmpx2 = (3.0 / 4.0) + (1.0 / 8.0) - coll0 / (8.0 * roomsize);

				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
		}

		if (checker == 0) {
			buffer[4 * tmp] = 0;
			buffer[4 * tmp + 1] = 0;
			buffer[4 * tmp + 2] = 0;

		}
		else
		{
			buffer[4 * tmp] = ctmp0;
			buffer[4 * tmp + 1] = ctmp1;
			buffer[4 * tmp + 2] = ctmp2;
		}
	}
}

void cudaInit()
{
	double dist = 2.0;
	double sqsz = 0.01 / 4;
	int tmpx, tmpy;
	double* vecltmp = new double[1920 * 1080];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double vecn0, vecn1, vecn2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1920) * sqsz / 2;
	double multz = (1080 - 1) * sqsz / 2;

	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1920 * 1080 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1920 * 1080 * sizeof(double));
	hipMalloc((void**)&stars, starsize * starsize * sizeof(uint8_t));
	
	vec0 = dist * x00 + multy * x10 + multz * x20;
	vec1 = dist * x01 + multy * x11 + multz * x21;
	vec2 = dist * x02 + multy * x12 + multz * x22;
	
	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;
	
	for (int i = 0; i < 1920 * 1080; i++)
	{
		tmpx = i % 1920;
		tmpy = (i - tmpx) / 1920;

		vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
		vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
		vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

		vecltmp[i] = sqrt(vecn0*vecn0+vecn1*vecn1+vecn2*vecn2);
	}

	hipMemcpy(vecl, vecltmp, 1920 * 1080 * sizeof(double), hipMemcpyHostToDevice);

	bufferinit << <(int)(1920 * 1080 / 600), 600 >> > (buffer);
	hipDeviceSynchronize();

	setstars << <starsize * starsize / 500, 500 >> > (stars);
	hipDeviceSynchronize();
}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, bool inside, double alpha, double beta, double bigr, double r, bool other)
{
	addKernel <<<(int)(1920 * 1080 / 600), 600>>>(buffer, vecl, pos0,pos1,pos2,vec0,vec1,vec2,addy0,addy1,addy2,addz0,addz1,addz2,inside,alpha, beta,bigr,  r,other,stars);

	hipDeviceSynchronize();
	hipMemcpy(pixels, buffer, 4 * 1920 * 1080 * sizeof(uint8_t), hipMemcpyDeviceToHost);
}
